#include "hip/hip_runtime.h"
#include "icc_cuda.h"
#include <thrust/version.h>
// #include <thrust/reduce.h>
// #include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
// #include <thrust/transform_reduce.h>
// #include <thrust/functional.h>
#include <thrust/sort.h>
// #include </usr/include/python2.6/Python.h>
// #include <boost/python.hpp>

#define chkErrq(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
  {
    cout << "GPUasser: " << hipGetErrorString(code) << "   "  << file << "  "  << line << endl;
    if (abort) exit(code);
  }
}

#define chkErrqCusparse(ans) { cusparseAssert((ans), __FILE__, __LINE__); }
inline void cusparseAssert(hipsparseStatus_t code, const char *file, int line, bool abort=true)
{
  if (code != 0) 
  {
    if(code == 1)
      cout << code << " cusparseStatusNotInitialized " << file << "  " << line << endl;
    else if(code == 2)
      cout << code << " cusparseStatusAllocFailed " << file << "  " << line << endl;
    else if (code == 3)
      cout << code << " cusparseStatusInvalidValue " << file << "  " << line << endl;
    else if (code == 4)
      cout << code << " cusparseStatusArchMismatch " << file << "  " << line << endl;
    else if (code == 5)
      cout << code << " cusparseStatusMappingError " << file << "  " << line << endl;
    else if (code == 6)
      cout << code << " cusparseStatusExecutionFailed " << file << "  " << line << endl;
    else if (code == 7)
      cout << code << " cusparseStatusInternalError " << file << "  " << line << endl;
    else if (code == 8)
      cout << code << " cusparseStatusMatrixTypeNotSupported " << file << "  " << line << endl;
    // cout << "cuSparseasser: " << code << "   "  << file << "  "  << line << endl;
    if (abort) exit(code);
  }
}

struct saxpy_functor
{
  const int m;
  saxpy_functor(int _m) : m(_m) {}

    __host__ __device__
    unsigned long long int operator()(const int& x, const unsigned long long int& y) const { 
      return m * x + y;
    }
};

void saxpy_fast(int m, thrust::device_vector<int>& X, thrust::device_vector<unsigned long long int>& Y)
{
  // Y <- m * X + Y
  thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), saxpy_functor(m));
}


/*
 mobilityUFRPY computes the 3x3 RPY mobility
 between blobs i and j normalized with 8 pi eta a
*/
__device__ void mobilityUFRPY(double rx,
			      double ry,
			      double rz,
			      double &Mxx,
			      double &Mxy,
			      double &Mxz,
			      double &Myy,
			      double &Myz,
			      double &Mzz,
			      int i,
			      int j,
                              double invaGPU){
  
  double fourOverThree = 4.0 / 3.0;

  if(i==j){
    Mxx = fourOverThree;
    Mxy = 0;
    Mxz = 0;
    Myy = Mxx;
    Myz = 0;
    Mzz = Mxx;
  }
  else{
    rx = rx * invaGPU; //Normalize distance with hydrodynamic radius
    ry = ry * invaGPU;
    rz = rz * invaGPU;
    double r2 = rx*rx + ry*ry + rz*rz;
    double r = sqrt(r2);
    //We should not divide by zero but std::numeric_limits<double>::min() does not work in the GPU
    //double invr = (r > std::numeric_limits<double>::min()) ? (1.0 / r) : (1.0 / std::numeric_limits<double>::min())
    double invr = 1.0 / r;
    double invr2 = invr * invr;
    double c1, c2;
    if(r>=2){
      c1 = 1 + 2 / (3 * r2);
      c2 = (1 - 2 * invr2) * invr2;
      Mxx = (c1 + c2*rx*rx) * invr;
      Mxy = (     c2*rx*ry) * invr;
      Mxz = (     c2*rx*rz) * invr;
      Myy = (c1 + c2*ry*ry) * invr;
      Myz = (     c2*ry*rz) * invr;
      Mzz = (c1 + c2*rz*rz) * invr;
    }
    else{
      c1 = fourOverThree * (1 - 0.28125 * r); // 9/32 = 0.28125
      c2 = fourOverThree * 0.09375 * invr;    // 3/32 = 0.09375
      Mxx = c1 + c2 * rx*rx ;
      Mxy =      c2 * rx*ry ;
      Mxz =      c2 * rx*rz ;
      Myy = c1 + c2 * ry*ry ;
      Myz =      c2 * ry*rz ;
      Mzz = c1 + c2 * rz*rz ;
    }
  } 
  return;
}


/*
 mobilityRPY computes the 3x3 mobility correction due to a wall
 between blobs i and j normalized with 8 pi eta a.
 This uses the expression from the Swan and Brady paper for a finite size particle.
 Mobility is normalize by 8*pi*eta*a.
*/
__device__ void mobilityUFSingleWallCorrection(double rx,
			                       double ry,
			                       double rz,
			                       double &Mxx,
                  			       double &Mxy,
			                       double &Mxz,
                                               double &Myx,
			                       double &Myy,
			                       double &Myz,
                                               double &Mzx,
                                               double &Mzy,
			                       double &Mzz,
			                       int i,
			                       int j,
                                               double invaGPU,
                                               double hj){
  if(i == j){
    double invZi = 1.0 / hj;
    Mxx += -(9*invZi - 2*pow(invZi,3) + pow(invZi,5)) / 12.0;
    Myy += -(9*invZi - 2*pow(invZi,3) + pow(invZi,5)) / 12.0;
    Mzz += -(9*invZi - 4*pow(invZi,3) + pow(invZi,5)) / 6.0;
  }
  else{
    double h_hat = hj / rz;
    double invR = rsqrt(rx*rx + ry*ry + rz*rz); // = 1 / r;
    double ex = rx * invR;
    double ey = ry * invR;
    double ez = rz * invR;
    
    double fact1 = -(3*(1+2*h_hat*(1-h_hat)*ez*ez) * invR + 2*(1-3*ez*ez) * pow(invR,3) - 2*(1-5*ez*ez) * pow(invR,5))  / 3.0;
    double fact2 = -(3*(1-6*h_hat*(1-h_hat)*ez*ez) * invR - 6*(1-5*ez*ez) * pow(invR,3) + 10*(1-7*ez*ez) * pow(invR,5)) / 3.0;
    double fact3 =  ez * (3*h_hat*(1-6*(1-h_hat)*ez*ez) * invR - 6*(1-5*ez*ez) * pow(invR,3) + 10*(2-7*ez*ez) * pow(invR,5)) * 2.0 / 3.0;
    double fact4 =  ez * (3*h_hat*invR - 10*pow(invR,5)) * 2.0 / 3.0;
    double fact5 = -(3*h_hat*h_hat*ez*ez*invR + 3*ez*ez*pow(invR, 3) + (2-15*ez*ez)*pow(invR, 5)) * 4.0 / 3.0;
    
    Mxx += fact1 + fact2 * ex*ex;
    Mxy += fact2 * ex*ey;
    Mxz += fact2 * ex*ez + fact3 * ex;
    Myx += fact2 * ey*ex;
    Myy += fact1 + fact2 * ey*ey;
    Myz += fact2 * ey*ez + fact3 * ey;
    Mzx += fact2 * ez*ex + fact4 * ex;
    Mzy += fact2 * ez*ey + fact4 * ey;
    Mzz += fact1 + fact2 * ez*ez + fact3 * ez + fact4 * ez + fact5;         
  }
}


/*
  Determine number of non-zero elements (nnz)
*/
__global__ void countNnz(const double *x, unsigned long long int *nnzGPU, const double cutoff, const int N){

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i >= N) return;   

  double rx, ry, rz, r2;
  int NDIM = 3; // 3 is the spatial dimension
  int ioffset = i * NDIM; 
  int joffset;
  
  // Loop over columns
  for(int j=0; j<N; j++){
    joffset = j * NDIM;
    
    // Compute vector between blobs i and j
    rx = x[ioffset    ] - x[joffset    ];
    ry = x[ioffset + 1] - x[joffset + 1];
    rz = x[ioffset + 2] - x[joffset + 2];
    r2 = (rx*rx + ry*ry + rz*rz);
    
    // If blobs are close increse nnz
    if(r2 < cutoff*cutoff){
      unsigned long long int nnz_old = atomicAdd(nnzGPU, 9);
    }
  }
}


/*
  Build a sparse matrix with coordinated format (COO). See cuSparse documentation.
*/
__global__ void buildCOOMatrix(const double *x,
			       double *cooValA,
                               int *cooRowIndA,
                               int *cooColIndA,
                               unsigned long long int *nnzGPU,
			       const double eta,
			       const double a,
			       const double cutoff,
			       const int N){

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i >= N) return;   

  double pi = 4.0 * atan(1.0);
  double norm_fact = 1.0 / (8 * pi * eta * a);  
  double inva = 1.0 / a;
  double rx, ry, rz, r2;
  int NDIM = 3; // 3 is the spatial dimension
  int ioffset = i * NDIM; 
  int joffset;
  double Mxx, Mxy, Mxz;
  double Myx, Myy, Myz;
  double Mzx, Mzy, Mzz;

  // Loop over columns
  for(int j=0; j<N; j++){
    joffset = j * NDIM;
    
    // Compute vector between blobs i and j
    rx = x[ioffset    ] - x[joffset    ];
    ry = x[ioffset + 1] - x[joffset + 1];
    rz = x[ioffset + 2] - x[joffset + 2];
    r2 = (rx*rx + ry*ry + rz*rz);
    
    // If blobs are close compute pair-mobility
    if(r2 < cutoff*cutoff){
      mobilityUFRPY(rx,ry,rz, Mxx,Mxy,Mxz,Myy,Myz,Mzz, i,j, inva);
      Myx = Mxy;
      Mzx = Mxz;
      Mzy = Myz;
      mobilityUFSingleWallCorrection(rx/a, ry/a, (rz+2*x[joffset+2])/a, Mxx,Mxy,Mxz,Myx,Myy,Myz,Mzx,Mzy,Mzz, i,j, inva, x[joffset+2]/a);
      
      int nnz_old = atomicAdd(nnzGPU, 9);      
      cooValA[nnz_old] = Mxx * norm_fact;
      cooRowIndA[nnz_old] = ioffset;
      cooColIndA[nnz_old] = joffset;

      nnz_old++;
      cooValA[nnz_old] = Mxy * norm_fact;
      cooRowIndA[nnz_old] = ioffset;
      cooColIndA[nnz_old] = joffset + 1;

      nnz_old++;
      cooValA[nnz_old] = Mxz * norm_fact;
      cooRowIndA[nnz_old] = ioffset;
      cooColIndA[nnz_old] = joffset + 2;

      nnz_old++;
      cooValA[nnz_old] = Myx * norm_fact;
      cooRowIndA[nnz_old] = ioffset + 1;
      cooColIndA[nnz_old] = joffset;

      nnz_old++;
      cooValA[nnz_old] = Myy * norm_fact;
      cooRowIndA[nnz_old] = ioffset + 1;
      cooColIndA[nnz_old] = joffset + 1;

      nnz_old++;
      cooValA[nnz_old] = Myz * norm_fact;
      cooRowIndA[nnz_old] = ioffset + 1;
      cooColIndA[nnz_old] = joffset + 2;

      nnz_old++;
      cooValA[nnz_old] = Mzx * norm_fact;
      cooRowIndA[nnz_old] = ioffset + 2;
      cooColIndA[nnz_old] = joffset ;

      nnz_old++;
      cooValA[nnz_old] = Mzy * norm_fact;
      cooRowIndA[nnz_old] = ioffset + 2;
      cooColIndA[nnz_old] = joffset + 1;

      nnz_old++;
      cooValA[nnz_old] = Mzz * norm_fact;
      cooRowIndA[nnz_old] = ioffset + 2;
      cooColIndA[nnz_old] = joffset + 2;
    } 
  }  
} 


/*
  Constructor: build the sparse mobility matrix M
  and compute the Cholesky factorization M=L*L.T
  where L is a lower triangular matrix.
*/
icc::icc(const double blob_radius, 
	 const double eta, 
	 const double cutoff,
	 const int number_of_blobs,
	 const double *x){
  d_blob_radius = blob_radius;
  d_eta = eta;
  d_cutoff = cutoff;
  d_number_of_blobs = number_of_blobs;
  d_x = x;

  // Determine number of blocks and threads for the GPU
  d_threads_per_block = 512;
  if((d_number_of_blobs / d_threads_per_block) < 512){
    d_threads_per_block = 256;
  }
  if((d_number_of_blobs / d_threads_per_block) < 256){
    d_threads_per_block = 128;
  }
  if((d_number_of_blobs / d_threads_per_block) < 128){
    d_threads_per_block = 128;
  }
  if((d_number_of_blobs / d_threads_per_block) < 128){
    d_threads_per_block = 64;
  }
  if((d_number_of_blobs / d_threads_per_block) < 32){
    d_threads_per_block = 128;
  }
  d_num_blocks = (d_number_of_blobs - 1) / d_threads_per_block + 1;
}

/*
  Destructor: free memory on the GPU and CPU.
*/
icc::~icc(){
  // Delete cusparse objects
  cout << "destroying " << endl;
  chkErrqCusparse(cusparseDestroySolveAnalysisInfo(d_info_M)); 
  cout << "vvvv" << endl;
  hipsparseDestroyMatDescr(d_descr_M);
  cout << "DDD" << endl;

  chkErrqCusparse(hipsparseDestroy(d_cusp_handle));
  cout << "AAA " << endl;

  // Free GPU memory
  chkErrq(hipFree(d_x_gpu));
  chkErrq(hipFree(d_nnz_gpu));
  chkErrq(hipFree(d_cooVal_gpu));
  chkErrq(hipFree(d_cooVal_sorted_gpu));
  chkErrq(hipFree(d_cooRowInd_gpu));
  chkErrq(hipFree(d_cooColInd_gpu));
  chkErrq(hipFree(d_csrRowPtr_gpu));
}

/*
  Build sparse mobility matrix M.
*/
int icc::buildSparseMobilityMatrix(){
  int N = d_number_of_blobs * 3;
  
  // Allocate GPU memory
  chkErrq(hipMalloc((void**)&d_x_gpu, N * sizeof(double)));
  chkErrq(hipMalloc((void**)&d_nnz_gpu, sizeof(unsigned long long int)));

  // Copy data from CPU to GPU
  chkErrq(hipMemcpy(d_x_gpu, d_x, N * sizeof(double), hipMemcpyHostToDevice));
  d_nnz = 0;
  chkErrq(hipMemcpy(d_nnz_gpu, &d_nnz, sizeof(unsigned long long int), hipMemcpyHostToDevice));

  // Count non-zero elements in mobility matrix
  countNnz<<<d_num_blocks, d_threads_per_block>>>(d_x_gpu, d_nnz_gpu, d_cutoff, d_number_of_blobs);
  chkErrq(hipPeekAtLastError());
  chkErrq(hipMemcpy(&d_nnz, d_nnz_gpu, sizeof(unsigned long long int), hipMemcpyDeviceToHost));
  cout << "nnz = " << d_nnz << endl;

  // Allocate GPU memory for the sparse mobility matrix
  chkErrq(hipMalloc((void**)&d_cooVal_gpu, d_nnz * sizeof(double)));
  chkErrq(hipMalloc((void**)&d_cooVal_sorted_gpu, d_nnz * sizeof(double)));
  chkErrq(hipMalloc((void**)&d_cooRowInd_gpu, d_nnz * sizeof(int)));
  chkErrq(hipMalloc((void**)&d_cooColInd_gpu, d_nnz * sizeof(int)));
  chkErrq(hipMalloc((void**)&d_csrRowPtr_gpu, ((3 * d_number_of_blobs) + 1) * sizeof(int)));

  // Build sparse mobility matrix
  d_nnz = 0;
  chkErrq(hipMemcpy(d_nnz_gpu, &d_nnz, sizeof(unsigned long long int), hipMemcpyHostToDevice));
  buildCOOMatrix<<<d_num_blocks, d_threads_per_block>>>(d_x_gpu,
							d_cooVal_gpu,
							d_cooRowInd_gpu,
							d_cooColInd_gpu,
							d_nnz_gpu,
							d_eta,
							d_blob_radius,
							d_cutoff,
							d_number_of_blobs);
  chkErrq(hipPeekAtLastError());
  chkErrq(hipMemcpy(&d_nnz, d_nnz_gpu, sizeof(unsigned long long int), hipMemcpyDeviceToHost));
  cout << "nnz = " << d_nnz << endl;
  
  // Init cuSparse
  chkErrqCusparse(hipsparseCreate(&d_cusp_handle));
  d_base = hipsparseIndexBase_t(0);

  // Sort matrix to COO format
  {
    thrust::device_vector<int> vec_col(d_cooColInd_gpu, d_cooColInd_gpu + d_nnz);
    thrust::device_vector<int> vec_row(d_cooRowInd_gpu, d_cooRowInd_gpu + d_nnz);
    thrust::device_vector<double> vec_val(d_cooVal_gpu, d_cooVal_gpu + d_nnz);
    thrust::device_vector<int> vec_col_sorted(d_nnz);
    thrust::device_vector<int> vec_row_sorted(d_nnz);
    thrust::device_vector<double> vec_val_sorted(d_nnz);
    thrust::device_vector<unsigned long long int> vec_global_index(d_cooColInd_gpu, d_cooColInd_gpu + d_nnz);

    if(0){
      cout << "Print values  ";
      thrust::copy(vec_val.begin(), vec_val.end(), std::ostream_iterator<double>(std::cout, " "));
      cout << endl;
      cout << "Print columns ";
      thrust::copy(vec_global_index.begin(), vec_global_index.end(), std::ostream_iterator<unsigned long long int>(std::cout, " "));
      cout << endl;
      cout << "Print rows    ";
      thrust::copy(vec_row.begin(), vec_row.end(), std::ostream_iterator<int>(std::cout, " "));
      cout << endl;
      // thrust::sort(d_cooRowInd, d_cooRowInd + d_nnz);
    }
    // Create global index = row*N + col
    saxpy_fast(N, vec_row, vec_global_index);
    if(0){
      // thrust::host_vector<unsigned long long int> vec_global_index_host = vec_global_index;
      cout << "Print index  ";
      thrust::copy(vec_global_index.begin(), vec_global_index.end(), std::ostream_iterator<unsigned long long int>(std::cout, " "));
      cout << endl;
    }

    // Initialize vector to [0, 1, 2, ...]
    thrust::counting_iterator<int> iter(0);
    thrust::device_vector<int> indices(d_nnz);
    thrust::copy(iter, iter + indices.size(), indices.begin());

    // Sort the indices using the global index as the key
    // thrust::sort_by_key(vec_index.begin()
    thrust::sort_by_key(vec_global_index.begin(), vec_global_index.end(), indices.begin());
    if(0){
      cout << "Print index  ";
      thrust::copy(vec_global_index.begin(), vec_global_index.end(), std::ostream_iterator<unsigned long long int>(std::cout, "  "));
      cout << endl;
    }

    // Sort rows, columns and values with the indices
    thrust::gather(indices.begin(), indices.end(), vec_col.begin(), vec_col_sorted.begin());
    thrust::gather(indices.begin(), indices.end(), vec_row.begin(), vec_row_sorted.begin());
    thrust::gather(indices.begin(), indices.end(), vec_val.begin(), vec_val_sorted.begin());

    if(0){
      cout << endl << endl << endl;
      cout << "Print columns ";
      thrust::copy(vec_col_sorted.begin(), vec_col_sorted.end(), std::ostream_iterator<int>(std::cout, " "));
      cout << endl;
      cout << "Print rows    ";
      thrust::copy(vec_row_sorted.begin(), vec_row_sorted.end(), std::ostream_iterator<int>(std::cout, " "));
      cout << endl;
      cout << "Print values  ";
      thrust::copy(vec_val_sorted.begin(), vec_val_sorted.end(), std::ostream_iterator<double>(std::cout, " "));
      cout << endl;
    }
    
    // Copy thrust vectors to arrays
    thrust::copy(vec_col_sorted.begin(), vec_col_sorted.end(), d_cooColInd_gpu);
    thrust::copy(vec_row_sorted.begin(), vec_row_sorted.end(), d_cooRowInd_gpu);
    thrust::copy(vec_val_sorted.begin(), vec_val_sorted.end(), d_cooVal_gpu);
  }

  // Transform sparse matrix to CSR format
  chkErrqCusparse(hipsparseXcoo2csr(d_cusp_handle, d_cooRowInd_gpu, d_nnz, N, d_csrRowPtr_gpu, d_base));
  
  // Copy matrix to the CPU
  if(0){
    d_cooVal = new double [d_nnz];
    d_cooRowInd = new int [d_nnz];
    d_cooColInd = new int [d_nnz];
    d_csrRowPtr = new int [(N) + 1];
    chkErrq(hipMemcpy(d_cooVal, d_cooVal_gpu, d_nnz * sizeof(double), hipMemcpyDeviceToHost));
    chkErrq(hipMemcpy(d_cooRowInd, d_cooRowInd_gpu, d_nnz * sizeof(int), hipMemcpyDeviceToHost));
    chkErrq(hipMemcpy(d_cooColInd, d_cooColInd_gpu, d_nnz * sizeof(int), hipMemcpyDeviceToHost));
    chkErrq(hipMemcpy(d_csrRowPtr, d_csrRowPtr_gpu, ((3 * d_number_of_blobs) + 1) * sizeof(int), hipMemcpyDeviceToHost));
    
    for(int i=0; i<d_nnz; i++){
      cout << i << " --- " << d_cooRowInd[i] << "  " << d_cooColInd[i] << "  " << d_cooVal[i] << endl;
    }
    for(int i=0; i < ((N) + 1); i++){
      cout << i << " --- " << d_csrRowPtr[i] << endl;
    }
    delete[] d_cooVal;
    delete[] d_cooRowInd;
    delete[] d_cooColInd;
    delete[] d_csrRowPtr;
  }

  // Create descriptor for matrix M
  chkErrqCusparse(hipsparseCreateMatDescr(&d_descr_M));
  chkErrqCusparse(hipsparseSetMatIndexBase(d_descr_M, HIPSPARSE_INDEX_BASE_ZERO));
  // chkErrqCusparse(hipsparseSetMatType(d_descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL));
  chkErrqCusparse(hipsparseSetMatType(d_descr_M, HIPSPARSE_MATRIX_TYPE_SYMMETRIC));
  chkErrqCusparse(hipsparseSetMatFillMode(d_descr_M, HIPSPARSE_FILL_MODE_LOWER));
  chkErrqCusparse(hipsparseSetMatDiagType(d_descr_M, HIPSPARSE_DIAG_TYPE_NON_UNIT));

  // Create info structure
  // hipsparseCreateCsric02Info(&d_info_M); for version 7.5
  cusparseCreateSolveAnalysisInfo(&d_info_M);
  hipsparseOperation_t operation = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  cout << "AAA " << endl;
  if(1){
    chkErrqCusparse(cusparseDcsrsv_analysis(d_cusp_handle, 
					    operation, /*HIPSPARSE_OPERATION_NON_TRANSPOSE*/
					    N,
					    d_nnz,
					    d_descr_M, 
					    d_cooVal_gpu,
					    d_csrRowPtr_gpu, 
					    d_cooColInd_gpu,
					    d_info_M));
    chkErrq(hipDeviceSynchronize());
  }

  // Compute incomplete cholesky 
  if(1){
    // chkErrqCusparse(hipsparseSetMatType(d_descr_M, HIPSPARSE_MATRIX_TYPE_SYMMETRIC));
    chkErrqCusparse(cusparseDcsric0(d_cusp_handle,
				    operation,
				    N,
				    d_descr_M,
				    d_cooVal_gpu,
				    d_csrRowPtr_gpu,
				    d_cooColInd_gpu,
				    d_info_M));
  }
  chkErrq(hipDeviceSynchronize());

    

  return 0;
}


int main(){

  // Define parameters
  int status;
  double blob_radius = 1.0;
  double eta = 1.0;
  double cutoff = 10;
  int number_of_blobs = 2;

  // Create CPU arrays
  double *x = new double [number_of_blobs * 3];
  for(int i=0; i<(number_of_blobs * 3); i++){
    x[i] = 10.0 * rand() / RAND_MAX;
    cout << i << "  " << x[i] << endl;
  }
 
  // Create icc object
  icc icc_obj = icc(blob_radius, eta, cutoff, number_of_blobs, x);
  
  // Build sparse mobility matrix
  status = icc_obj.buildSparseMobilityMatrix();
  cout << "Build sparse mobility matrix = " << status << endl;
  

  // Free CPU memory
  cout << "before x" << endl;
  delete[] x;
  cout << "# End" << endl;
  return 0;
}
